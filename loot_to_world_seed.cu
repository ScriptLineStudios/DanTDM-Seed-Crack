
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>

#ifndef RNG_H_
#define RNG_H_

#define __STDC_FORMAT_MACROS 1

#include <stdlib.h>
#include <stddef.h>
#include <inttypes.h>


///=============================================================================
///                      Compiler and Platform Features
///=============================================================================

typedef int8_t      i8;
typedef uint8_t     u8;
typedef int16_t     i16;
typedef uint16_t    u16;
typedef int32_t     i32;
typedef uint32_t    u32;
typedef int64_t     i64;
typedef uint64_t    u64;
typedef float       f32;
typedef double      f64;

#define XRSR_MIX1          0xbf58476d1ce4e5b9
#define XRSR_MIX2          0x94d049bb133111eb
#define XRSR_MIX1_INVERSE  0x96de1b173f119089
#define XRSR_MIX2_INVERSE  0x319642b2d24d8ec3
#define XRSR_SILVER_RATIO  0x6a09e667f3bcc909
#define XRSR_GOLDEN_RATIO  0x9e3779b97f4a7c15

__device__ __host__ uint64_t mix64(uint64_t a) {
	a = (a ^ a >> 30) * XRSR_MIX1;
	a = (a ^ a >> 27) * XRSR_MIX2;
	return a ^ a >> 31;
}

#define STRUCT(S) typedef struct S S; struct S

#if __GNUC__

#define IABS(X)                 __builtin_abs(X)
#define PREFETCH(PTR,RW,LOC)    __builtin_prefetch(PTR,RW,LOC)
#define likely(COND)            (__builtin_expect(!!(COND),1))
#define unlikely(COND)          (__builtin_expect((COND),0))
#define ATTR(...)               __attribute__((__VA_ARGS__))
#define BSWAP32(X)              __builtin_bswap32(X)
#define UNREACHABLE()           __builtin_unreachable()

#else

#define IABS(X)                 ((int)abs(X))
#define PREFETCH(PTR,RW,LOC)
#define likely(COND)            (COND)
#define unlikely(COND)          (COND)
#define ATTR(...)
static inline uint32_t BSWAP32(uint32_t x) {
    x = ((x & 0x000000ff) << 24) | ((x & 0x0000ff00) <<  8) |
        ((x & 0x00ff0000) >>  8) | ((x & 0xff000000) >> 24);
    return x;
}
#if _MSC_VER
#define UNREACHABLE()           __assume(0)
#else
#define UNREACHABLE()           exit(1) // [[noreturn]]
#endif

#endif

/// imitate amd64/x64 rotate instructions

static inline ATTR(const, always_inline, artificial)
__device__ __host__ uint64_t rotl64(uint64_t x, uint8_t b)
{
    return (x << b) | (x >> (64-b));
}

static inline ATTR(const, always_inline, artificial)
__device__ __host__ uint32_t rotr32(uint32_t a, uint8_t b)
{
    return (a >> b) | (a << (32-b));
}

/// integer floor divide
static inline ATTR(const, always_inline)
int32_t floordiv(int32_t a, int32_t b)
{
    int32_t q = a / b;
    int32_t r = a % b;
    return q - ((a ^ b) < 0 && !!r);
}

///=============================================================================
///                    C implementation of Java Random
///=============================================================================

__device__ __host__ static inline void setSeed(uint64_t *seed, uint64_t value)
{
    *seed = (value ^ 0x5deece66d) & ((1ULL << 48) - 1);
}

__device__ __host__ static inline int next(uint64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1ULL << 48) - 1);
    return (int) ((int64_t)*seed >> (48 - bits));
}

__device__ __host__ static inline int nextInt(uint64_t *seed, const int n) {
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (uint64_t)next(seed, 31);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

__device__ __host__ static inline int nextIntBounded(uint64_t *seed, const int min, const int max) {
    if (min >= max) {
        return min;
    }
    return nextInt(seed, max - min + 1) + min;
}

__device__ __host__ static inline uint64_t nextLong(uint64_t *seed)
{
    return ((uint64_t) next(seed, 32) << 32) + next(seed, 32);
}

__device__ __host__ static inline float nextFloat(uint64_t *seed)
{
    return next(seed, 24) / (float) (1 << 24);
}

__device__ __host__ static inline double nextDouble(uint64_t *seed)
{
    uint64_t x = (uint64_t)next(seed, 26);
    x <<= 27;
    x += next(seed, 27);
    return (int64_t) x / (double) (1ULL << 53);
}

/* A macro to generate the ideal assembly for X = nextInt(*S, 24)
 * This is a macro and not an inline function, as many compilers can make use
 * of the additional optimisation passes for the surrounding code.
 */
#define JAVA_NEXT_INT24(S,X)                \
    do {                                    \
        uint64_t a = (1ULL << 48) - 1;      \
        uint64_t c = 0x5deece66dULL * (S);  \
        c += 11; a &= c;                    \
        (S) = a;                            \
        a = (uint64_t) ((int64_t)a >> 17);  \
        c = 0xaaaaaaab * a;                 \
        c = (uint64_t) ((int64_t)c >> 36);  \
        (X) = (int)a - (int)(c << 3) * 3;   \
    } while (0)


/* Jumps forwards in the random number sequence by simulating 'n' calls to next.
 */
static inline void skipNextN(uint64_t *seed, uint64_t n)
{
    uint64_t m = 1;
    uint64_t a = 0;
    uint64_t im = 0x5deece66dULL;
    uint64_t ia = 0xb;
    uint64_t k;

    for (k = n; k; k >>= 1)
    {
        if (k & 1)
        {
            m *= im;
            a = im * a + ia;
        }
        ia = (im + 1) * ia;
        im *= im;
    }

    *seed = *seed * m + a;
    *seed &= 0xffffffffffffULL;
}


///=============================================================================
///                               Xoroshiro 128
///=============================================================================

STRUCT(Xoroshiro)
{
    uint64_t lo, hi;
};

__device__ __host__ static inline void xSetSeed(Xoroshiro *xr, uint64_t value)
{
    const uint64_t XL = 0x9e3779b97f4a7c15ULL;
    const uint64_t XH = 0x6a09e667f3bcc909ULL;
    const uint64_t A = 0xbf58476d1ce4e5b9ULL;
    const uint64_t B = 0x94d049bb133111ebULL;
    uint64_t l = value ^ XH;
    uint64_t h = l + XL;
    l = (l ^ (l >> 30)) * A;
    h = (h ^ (h >> 30)) * A;
    l = (l ^ (l >> 27)) * B;
    h = (h ^ (h >> 27)) * B;
    l = l ^ (l >> 31);
    h = h ^ (h >> 31);
    xr->lo = l;
    xr->hi = h;
}

__device__ __host__ static inline void xSetFeatureSeed(Xoroshiro *xr, uint64_t p_190065_, int p_190066_, int p_190067_) {
    uint64_t i = p_190065_ + (long)p_190066_ + (long)(10000 * p_190067_);
    xSetSeed(xr, i);
}

__device__ __host__ static inline uint64_t xNextLong(Xoroshiro *xr)
{
    uint64_t l = xr->lo;
    uint64_t h = xr->hi;
    uint64_t n = rotl64(l + h, 17) + l;
    h ^= l;
    xr->lo = rotl64(l, 49) ^ h ^ (h << 21);
    xr->hi = rotl64(h, 28);
    return n;
}

__device__ __host__ static inline uint64_t xSetDecorationSeed(Xoroshiro *xr, uint64_t p_64691_, int p_64692_, int p_64693_) {
    // this.setSeed(p_64691_);
    xSetSeed(xr, p_64691_);
    uint64_t i = xNextLong(xr) | 1L;
    uint64_t j = xNextLong(xr) | 1L;
    uint64_t k = (uint64_t)p_64692_ * i + (uint64_t)p_64693_ * j ^ p_64691_;
    // this.setSeed(k);
    xSetSeed(xr, k);
    return k;
}

__device__ __host__ static inline int xNextInt(Xoroshiro *xr, uint32_t n)
{
    uint64_t r = (xNextLong(xr) & 0xFFFFFFFF) * n;
    if ((uint32_t)r < n)
    {
        while ((uint32_t)r < (~n + 1) % n)
        {
            r = (xNextLong(xr) & 0xFFFFFFFF) * n;
        }
    }
    return r >> 32;
}

__device__ __host__ static inline double xNextDouble(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-53)) * 1.1102230246251565E-16;
}

__device__ __host__ static inline float xNextFloat(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-24)) * 5.9604645E-8F;
}

__device__ __host__ static inline void xSkipN(Xoroshiro *xr, int count)
{
    while (count --> 0)
        xNextLong(xr);
}

__device__ __host__ static inline uint64_t xNextLongJ(Xoroshiro *xr)
{
    int32_t a = xNextLong(xr) >> 32;
    int32_t b = xNextLong(xr) >> 32;
    return ((uint64_t)a << 32) + b;
}

__device__ __host__ static inline int xNextIntJ(Xoroshiro *xr, uint32_t n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (xNextLong(xr) >> 33);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = (xNextLong(xr) >> 33);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}


//==============================================================================
//                              MC Seed Helpers
//==============================================================================

/**
 * The seed pipeline:
 *
 * getLayerSalt(n)                -> layerSalt (ls)
 * layerSalt (ls), worldSeed (ws) -> startSalt (st), startSeed (ss)
 * startSeed (ss), coords (x,z)   -> chunkSeed (cs)
 *
 * The chunkSeed alone is enough to generate the first PRNG integer with:
 *   mcFirstInt(cs, mod)
 * subsequent PRNG integers are generated by stepping the chunkSeed forwards,
 * salted with startSalt:
 *   cs_next = mcStepSeed(cs, st)
 */

static inline uint64_t mcStepSeed(uint64_t s, uint64_t salt)
{
    return s * (s * 6364136223846793005ULL + 1442695040888963407ULL) + salt;
}

static inline int mcFirstInt(uint64_t s, int mod)
{
    int ret = (int)(((int64_t)s >> 24) % mod);
    if (ret < 0)
        ret += mod;
    return ret;
}

static inline int mcFirstIsZero(uint64_t s, int mod)
{
    return (int)(((int64_t)s >> 24) % mod) == 0;
}

static inline uint64_t getChunkSeed(uint64_t ss, int x, int z)
{
    uint64_t cs = ss + x;
    cs = mcStepSeed(cs, z);
    cs = mcStepSeed(cs, x);
    cs = mcStepSeed(cs, z);
    return cs;
}

static inline uint64_t getLayerSalt(uint64_t salt)
{
    uint64_t ls = mcStepSeed(salt, salt);
    ls = mcStepSeed(ls, salt);
    ls = mcStepSeed(ls, salt);
    return ls;
}

static inline uint64_t getStartSalt(uint64_t ws, uint64_t ls)
{
    uint64_t st = ws;
    st = mcStepSeed(st, ls);
    st = mcStepSeed(st, ls);
    st = mcStepSeed(st, ls);
    return st;
}

static inline uint64_t getStartSeed(uint64_t ws, uint64_t ls)
{
    uint64_t ss = ws;
    ss = getStartSalt(ss, ls);
    ss = mcStepSeed(ss, 0);
    return ss;
}


///============================================================================
///                               Arithmatic
///============================================================================


/* Linear interpolations
 */
__device__ __host__ static inline double lerp(double part, double from, double to)
{
    return from + part * (to - from);
}

__device__ __host__ static inline double lerp2(
        double dx, double dy, double v00, double v10, double v01, double v11)
{
    return lerp(dy, lerp(dx, v00, v10), lerp(dx, v01, v11));
}

__device__ __host__ static inline double lerp3(
        double dx, double dy, double dz,
        double v000, double v100, double v010, double v110,
        double v001, double v101, double v011, double v111)
{
    v000 = lerp2(dx, dy, v000, v100, v010, v110);
    v001 = lerp2(dx, dy, v001, v101, v011, v111);
    return lerp(dz, v000, v001);
}

__device__ __host__ static inline double clampedLerp(double part, double from, double to)
{
    if (part <= 0) return from;
    if (part >= 1) return to;
    return lerp(part, from, to);
}

/* Find the modular inverse: (1/x) | mod m.
 * Assumes x and m are positive (less than 2^63), co-prime.
 */
static inline ATTR(const)
__device__ __host__ uint64_t mulInv(uint64_t x, uint64_t m)
{
    uint64_t t, q, a, b, n;
    if ((int64_t)m <= 1)
        return 0; // no solution

    n = m;
    a = 0; b = 1;

    while ((int64_t)x > 1)
    {
        if (m == 0)
            return 0; // x and m are co-prime
        q = x / m;
        t = m; m = x % m;     x = t;
        t = a; a = b - q * a; b = t;
    }

    if ((int64_t)b < 0)
        b += n;
    return b;
}


typedef struct {
    Xoroshiro internal;
} RNG; // Bruh I really didn't want to have to do this.

__device__ __host__ RNG rng_new() {
    return (RNG){.internal=(Xoroshiro){0}};
}

__device__ __host__ static inline void rng_set_seed(RNG *rng, uint64_t seed) {
    seed ^= XRSR_SILVER_RATIO;
    rng->internal.lo = mix64(seed);
    rng->internal.hi = mix64(seed + XRSR_GOLDEN_RATIO);
}

__device__ __host__ static inline void rng_set_internal(RNG *rng, uint64_t lo, uint64_t hi) {
    rng->internal.lo = lo;
    rng->internal.hi = hi;
}

__device__ __host__ static inline uint32_t rng_next(RNG *rng, int32_t bits) {
    return xNextLong(&rng->internal) >> (64 - bits);
}

__device__ __host__ static inline int32_t rng_next_int(RNG *rng, uint32_t bound) {
    uint32_t r = rng_next(rng, 31);
    uint32_t m = bound - 1;
    if ((bound & m) == 0) {
        // (int)((long)p_188504_ * (long)this.next(31) >> 31);
        r = (uint32_t)((uint64_t)bound * (uint64_t)r >> 31);
    }
    else {
        for (uint32_t u = r; (int32_t)(u - (r = u % bound) + m) < 0; u = rng_next(rng, 31));
    }
    return r;
}

__device__ __host__ static inline uint64_t rng_next_long(RNG *rng) {
    int32_t i = rng_next(rng, 32);
    int32_t j = rng_next(rng, 32);
    uint64_t k = (uint64_t)i << 32;
    return k + (uint64_t)j;
}

__device__ __host__ static inline uint64_t rng_set_decoration_seed(RNG *rng, uint64_t world_seed, int32_t x, int32_t z) {
    rng_set_seed(rng, world_seed);

    uint64_t a = rng_next_long(rng) | 1L;
    uint64_t b = rng_next_long(rng) | 1L;

    uint64_t k = (a * (uint64_t)x + b * (uint64_t)z) ^ world_seed;
    rng_set_seed(rng, k);
    return k;
}

__device__ __host__ static inline void rng_set_feature_seed(RNG *rng, uint64_t p_190065_, int32_t p_190066_, int32_t p_190067_) {
    uint64_t i = p_190065_ + (uint64_t)p_190066_ + (uint64_t)(10000 * p_190067_);
    //printf("Salt = %" PRIu64 "\n", (uint64_t)p_190066_ + (uint64_t)(10000 * p_190067_));
    rng_set_seed(rng, i);
}

#endif /* RNG_H_ */

#define ll  long long int

#define printu64(val) printf("%" PRIu64 "\n", (val))
#define printi64(val) printf("%" PRIi64 "\n", (val))
#define printi(val) printf("%d\n", (val))
#define print_seed(val) printi64(val)

__device__ const static uint64_t valid_loot_seeds[68] = {
    4835198300983L,
    7332035751452L,
    14125018631592L,
    16843522132883L,
    22558606346032L,
    25603415556092L,
    27917408974186L,
    28780665201524L,
    36763672506642L,
    49898174216347L,
    54851118410986L,
    67429533035729L,
    68921484724337L,
    79541903247155L,
    83760782148839L,
    83841116665372L,
    84176774411871L,
    90797196476473L,
    91357893484262L,
    92571097990103L,
    94798470879408L,
    96960812053490L,
    101164170990904L,
    101235091411158L,
    106565953754466L,
    106921894927966L,
    107365059663880L,
    111512993735855L,
    115233097849165L,
    116230392124471L,
    118051218661506L,
    118109354894248L,
    122663339622580L,
    123764680956139L,
    126157375814142L,
    128872024651236L,
    128896829384277L,
    130556325114174L,
    140455946282600L,
    143000543024508L,
    152240665975058L,
    154690659645272L,
    158158777792049L,
    164936178522953L,
    178065525828803L,
    183823501087772L,
    186275823599766L,
    187078375095561L,
    188030536518365L,
    196793879718854L,
    201845975496614L,
    204573765959703L,
    208270839479119L,
    209156717123810L,
    211393967678402L,
    215988698279009L,
    225715937633625L,
    226717405268749L,
    234029363196676L,
    237623813325737L,
    237888192141868L,
    253892783093514L,
    255547815557168L,
    258960756868937L,
    261560426974177L,
    269655404299125L,
    272605054575375L,
    280486279432499L
};

#define MASK48 0xFFFFFFFFFFFFULL
__managed__ unsigned long long seedsChecked = 0;

__global__ void kernel(uint64_t o)
{
    uint64_t input_seed = blockDim.x * blockIdx.x + threadIdx.x + o;

    uint64_t seed;
    setSeed(&seed, input_seed);
    uint64_t world_seed = nextLong(&seed);

    RNG rng = rng_new();

    uint64_t i = rng_set_decoration_seed(&rng, world_seed, 192, 0);
    rng_set_feature_seed(&rng, i, 10, 4);

    uint64_t loot_seed = rng_next_long(&rng);

    for (int i = 0; i < 1; i++) {
        if ((loot_seed & MASK48) == valid_loot_seeds[i]) {
            print_seed(world_seed);
        }
    }
    atomicAdd(&seedsChecked, 1);
}

int main(int argc,char **argv)
{
    uint64_t threads_per_block = 512L;
    uint64_t num_blocks = 32768L;

    printf("CPU: Hello!\n");
    const uint64_t max = threads_per_block * num_blocks * 100000L; //(1ll << 48);
	  for (ll o = 0; o < max; o += threads_per_block * num_blocks) {
      //printf("%lld %lld\n", o, max);
		  kernel<<<num_blocks, threads_per_block>>>(o);
    }
    hipDeviceSynchronize();
    printf("%lld\n", seedsChecked);
    return 0;
}